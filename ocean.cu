///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}

// complex math functions
__device__
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__
float2 complex_add(float2 a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}
__device__
float2 scalarMult(float2 a, float b)
{
    return make_float2(a.x * b, a.y *b);
}

__device__
float2 interp2F2(float2 a, float2 b, float d)
{
	return make_float2(a.x + d*(b.x-a.x), a.y + d*(b.y-a.y)); 
}
__device__
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

__device__
float absSqrt(float a, float thresh){
	if(a > thresh){	return sqrtf(a);	}
	else if (a < -thresh) {return -sqrtf(-a);}
}

//convert passed list of frequencies to appropriate array of float2
extern "C"
__global__ void buildFrequencyDataKernel(float2* freq_out,
										float* freq_rList,						//single dimension array of 1024 elements
										float* freq_cList,
                                       	unsigned int in_width,
                                       	unsigned int out_width,
                                       	unsigned int out_height,
										unsigned int is_NoteFreqs, 
									//	float thresh,
										float t){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    unsigned int out_index = y*out_width+x;
    unsigned int halfWidth = in_width/2;
    unsigned int inx = ((x+halfWidth) % (in_width))+1;
	unsigned int iny = in_width -(((y+halfWidth) % (in_width))+1); //mirrored

    float u = x / (float) out_width;
    float v = y / (float) out_height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;
    
	float scFct = .1f;
	t = t+scFct;
//	unsigned int totalOut = out_width * out_height;
//	unsigned int colOff = out_width/2;
//	unsigned int rowOff = (out_width * colOff);
//	unsigned int newIdx = (rowOff + (out_width*(out_index+colOff)/out_width) + 			
//				((colOff + (out_index%out_width)) % out_height))%totalOut;
	//if note frequencies, get complex version of note data, otherwise use freq_rList and freq_cList
	//e^j2pifot = cos(2pifot)<---freq_rList from audio + j(sin(2pifot) <---freq_cList from audio)    
	
//	if(is_NoteFreqs == 0){
		if ((x < out_width) && (y < out_height)) { 	//in_width == out_width
			float freqR = freq_rList[inx];
			float freqC = freq_cList[iny];
			//this squaring these maximizes the value at the center of the frequency map - this is where the philips noise value is
			float val =  (freqR  * freqC * 100) ;
	    	freq_out[out_index] = make_float2(val,val);
		}
	
//	} else {
//		if ((x < out_width) && (y < out_height)) { 	//need to send in FFT!
//			float freqR = (freq_rList[inx] < thresh ? thresh : freq_rList[inx]);
//			float freqC = (freq_cList[iny] < thresh ? thresh : freq_cList[iny]);
//			freqR = freqR / powf(2,llrintf(log2f(freqR+1))-1);
//			freqC = freqC / powf(2,llrintf(log2f(freqC+1))-1);
//			freq_out[out_index] = make_float2(sinf(u*freqR + t) * cosf(v*freqR + t) * scFct, sinf(v*freqC + t) * cosf(u*freqC + t) * scFct);
//		}
//	}
//	//freq_out[out_index] 

}
//convert passed list of frequencies to appropriate array of float2 - use sqrt
extern "C"
__global__ void buildFrequencyDataKernel2(float2* freq_out,
										float* freq_rList,						//single dimension array of 1024 elements
										float* freq_cList,
                                       	unsigned int in_width,
                                       	unsigned int out_width,
                                       	unsigned int out_height,
										unsigned int is_NoteFreqs,
									//	float thresh,
										float t){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int out_index = y*out_width+x;
    unsigned int halfWidth = in_width/2;
    unsigned int inx = ((x+halfWidth) % (in_width))+1;
	unsigned int iny = in_width -(((y+halfWidth) % (in_width))+1); //mirrored

    float u = x / (float) out_width;
    float v = y / (float) out_height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

	float scFct = .1f;
	t = t+scFct;
//	unsigned int totalOut = out_width * out_height;
//	unsigned int colOff = out_width/2;
//	unsigned int rowOff = (out_width * colOff);
//	unsigned int newIdx = (rowOff + (out_width*(out_index+colOff)/out_width) +
//				((colOff + (out_index%out_width)) % out_height))%totalOut;
	//if note frequencies, get complex version of note data, otherwise use freq_rList and freq_cList
	//e^j2pifot = cos(2pifot)<---freq_rList from audio + j(sin(2pifot) <---freq_cList from audio)

//	if(is_NoteFreqs == 0){
		if ((x < out_width) && (y < out_height)) { 	//in_width == out_width
			float freqR = freq_rList[inx];
			float freqC = freq_cList[iny];
			//this maximizes the value at the center of the frequency map - this is where the philips noise value is
			float val =  (freqR  * freqC * 10) ;
			val = absSqrt(val, 1);
	    	freq_out[out_index] = make_float2(val,val);
		}

//	} else {
//		if ((x < out_width) && (y < out_height)) { 	//need to send in FFT!
//			float freqR = (freq_rList[inx] < thresh ? thresh : freq_rList[inx]);
//			float freqC = (freq_cList[iny] < thresh ? thresh : freq_cList[iny]);
//			freqR = freqR / powf(2,llrintf(log2f(freqR+1))-1);
//			freqC = freqC / powf(2,llrintf(log2f(freqC+1))-1);
//			freq_out[out_index] = make_float2(sinf(u*freqR + t) * cosf(v*freqR + t) * scFct, sinf(v*freqC + t) * cosf(u*freqC + t) * scFct);
//		}
//	}
//	//freq_out[out_index]

}
// generate wave heightfield at time t based on initial heightfield and dispersion relationship : interp between noise and music
extern "C"
__global__ void generateSpectrumKernel(float2* h0, float2* ht,float2* freq, unsigned int in_width, unsigned int out_width, unsigned int out_height,
                                       float t,float mix,float patchSize){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;
    
    // calculate wave vector
    float2 k;
    float twoPiInvPtch = (2.0f * HIP_PI_F / patchSize);
    k.x = (-(int)out_width / 2.0f + x) * twoPiInvPtch;
    k.y = (-(int)out_height / 2.0f + y) * twoPiInvPtch;

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);
    float2 cmplxExp = complex_exp(w * t);
    float2 cmplxNExp = complex_exp(-w * t);

	if ((x < out_width) && (y < out_height)) {
		float2 h0_k = h0[in_index];
		float2 h0_mk = h0[in_mindex];
		float2 f0_k = freq[in_index];
		float2 f0_mk = freq[in_mindex];
		float2 tmpRes1 = complex_add( complex_mult(h0_k, cmplxExp), complex_mult(conjugate(h0_mk), cmplxNExp) );
		float2 tmpRes2 = complex_mult( tmpRes1, complex_add( complex_mult(f0_k, cmplxExp), complex_mult(conjugate(f0_mk), cmplxNExp) ));
		 // output frequency-space complex values
		ht[out_index] = interp2F2(tmpRes1,tmpRes2,mix);
	}
}
// generate wave heightfield at time t based on initial heightfield and dispersion relationship : interpolate between noise and music convolved with noise
extern "C"
__global__ void generateSpectrumKernel2(float2* h0, float2* ht,float2* freq, unsigned int in_width, unsigned int out_width, unsigned int out_height,
                                       float t,float mix,float patchSize){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;

    // calculate wave vector
    float2 k;
    float twoPiInvPtch = (2.0f * HIP_PI_F / patchSize);
    k.x = (-(int)out_width / 2.0f + x) * twoPiInvPtch;
    k.y = (-(int)out_height / 2.0f + y) * twoPiInvPtch;

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);
    float2 cmplxExp = complex_exp(w * t);
    float2 cmplxNExp = complex_exp(-w * t);

	if ((x < out_width) && (y < out_height)) {
		float2 h0_k = h0[in_index];
		float2 h0_mk = h0[in_mindex];
		float2 f0_k = freq[in_index];
		float2 f0_mk = freq[in_mindex];
		float2 tmpRes1 = complex_add( complex_mult(h0_k, cmplxExp), complex_mult(conjugate(h0_mk), cmplxNExp) );
		//set "wet" mix to be convolved noise with audio frequencies
		float2 tmpRes2 = scalarMult(complex_add( complex_mult(f0_k, cmplxExp), complex_mult(conjugate(f0_mk), cmplxNExp) ), .1f);
        // output frequency-space complex values
		ht[out_index] = interp2F2(tmpRes1,tmpRes2,mix);
	}
}

// update height map values based on output of FFT
extern "C"
__global__ void updateHeightmapKernel(float*  heightMap, float2* ht, unsigned int width){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;
    
    float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;
	heightMap[i] = ht[i].x * sign_correction;
}

// generate slope by partial differences in spatial domain
extern "C"
__global__ void calculateSlopeKernel(float* h, float2 *slopeOut, unsigned int width, unsigned int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float2 slope = make_float2(0.0f, 0.0f);
    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
        slope.x = h[i+1] - h[i-1];
        slope.y = h[i+width] - h[i-width];
    }
    slopeOut[i] = slope;
}
